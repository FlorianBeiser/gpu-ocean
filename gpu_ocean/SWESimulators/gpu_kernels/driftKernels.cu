
#include <hip/hip_runtime.h>
/*
This software is part of GPU Ocean. 

Copyright (C) 2018 SINTEF Digital
Copyright (C) 2018 Norwegian Meteorological Institute

This CUDA kernel implements a selection of drift trajectory algorithms.

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



/**
  * Kernel that evolves drifter positions along u and v.
  */
extern "C" {
__global__ void passiveDrifterKernel(
        //Discretization parameters
        int nx_, int ny_,
        float dx_, float dy_, float dt_,

	int x_zero_reference_cell_, // the cell column representing x0 (x0 at western face)
	int y_zero_reference_cell_, // the cell row representing y0 (y0 at southern face)
	
	// Data
        float* eta_ptr_, int eta_pitch_,
        float* hu_ptr_, int hu_pitch_,
        float* hv_ptr_, int hv_pitch_,
	// H should be read from buffer, but for now we use a constant value
	//__global float* H_ptr_, int H_pitch_,
	float H_,

	int periodic_north_south_,
	int periodic_east_west_,
	
	int num_drifters_,
	float* drifters_positions_, int drifters_pitch_,
	float sensitivity_
    ) {

    //Index of thread within block (only needed in one dim)
    const int tx = threadIdx.x;
        
    //Index of block within domain (only needed in one dim)
    const int bx = blockDim.x * blockIdx.x;
        
    //Index of cell within domain (only needed in one dim)
    const int ti = bx + tx;
    
    if (ti < num_drifters_ + 1) {
	// Obtain pointer to our particle:
	float* drifter = (float*) ((char*) drifters_positions_ + drifters_pitch_*ti);
	float drifter_pos_x = drifter[0];
	float drifter_pos_y = drifter[1];

	// Find cell ID for the cell in which our particle is
	int const cell_id_x = (int)(ceil(drifter_pos_x/dx_) + x_zero_reference_cell_);
	int const cell_id_y = (int)(ceil(drifter_pos_y/dy_) + y_zero_reference_cell_);

	// Read the water velocity from global memory
	float* const eta_row = (float*) ((char*) eta_ptr_ + eta_pitch_*cell_id_y);
	float const h = H_ + eta_row[cell_id_x];

	float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*cell_id_y);
	float const u = hu_row[cell_id_x]/h;

	float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*cell_id_y);
	float const v = hv_row[cell_id_x]/h;

        // Move drifter
	drifter_pos_x += sensitivity_*u*dt_;
	drifter_pos_y += sensitivity_*v*dt_;
        
        // Ensure boundary conditions
	if (periodic_east_west_ && (drifter_pos_x < 0)) {
	    drifter_pos_x += + nx_*dx_;
	}
	if (periodic_east_west_ && (drifter_pos_x > nx_*dx_)) {
	    drifter_pos_x -= nx_*dx_;
	}
	if (periodic_north_south_ && (drifter_pos_y < 0)) {
	    drifter_pos_y += ny_*dy_;
	}
	if (periodic_north_south_ && (drifter_pos_y > ny_*dy_)) {
	    drifter_pos_y -= ny_*dy_;
	}

	// Write to global memory
	drifter[0] = drifter_pos_x;
	drifter[1] = drifter_pos_y;
    }
}
} // extern "C"
    

extern "C" {
__global__ void enforceBoundaryConditions(
        //domain parameters
	float domain_size_x_, float domain_size_y_,

	int periodic_north_south_,
	int periodic_east_west_,
	
	int num_drifters_,
	float* drifters_positions_, int drifters_pitch_
    ) {
    
    //Index of drifter (only needed in one dimension)
    const int ti = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ti < num_drifters_ + 1) {
	// Obtain pointer to our particle:
	float* drifter = (float*) ((char*) drifters_positions_ + drifters_pitch_*ti);
	float drifter_pos_x = drifter[0];
	float drifter_pos_y = drifter[1];

	// Ensure boundary conditions
	if (periodic_east_west_ && (drifter_pos_x < 0)) {
	    drifter_pos_x += + domain_size_x_;
	}
	if (periodic_east_west_ && (drifter_pos_x > domain_size_x_)) {
	    drifter_pos_x -= domain_size_x_;
	}
	if (periodic_north_south_ && (drifter_pos_y < 0)) {
	    drifter_pos_y += domain_size_y_;
	}
	if (periodic_north_south_ && (drifter_pos_y > domain_size_y_)) {
	    drifter_pos_y -= domain_size_y_;
	}

	// Write to global memory
	drifter[0] = drifter_pos_x;
	drifter[1] = drifter_pos_y;
    }
}
} // extern "C"
